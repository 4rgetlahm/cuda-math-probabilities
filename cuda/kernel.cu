﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdio.h>

/* we need these includes for CUDA's random number stuff */
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>


#define ATTEMPT_COUNT 100000

/* 
#define N 5
#define M 13
*/

//GPU kernel for generating answer to probability problem where you need to find if there will be any bus stops that won't have any people leaving the bus
/*__global__ void generate_random_numbers(int max, unsigned long long* result) {

    int tId = threadIdx.x + (blockIdx.x * blockDim.x);

    curandState state;
    curand_init((unsigned long long)clock() + tId, 0, 0, &state);

    short stops[N] = {0};

    for (short i = 0; i < M; i++) {
        unsigned long long rand_number = ceilf(curand_uniform(&state) * max);
        stops[rand_number-1]++;
    }

    short count = 0;

    for (short i = 0; i < N; i++) {
        if (stops[i] == 0) {
            ++count;
        }
    }
    if (count == 2) {
        atomicAdd(result, 1);
        return;
    }
}*/

/*
#define FIRST_SHOOTER_CHANCE 0.42
#define SECOND_SHOOTER_CHANCE 0.34

__global__ void generate_shooting_range_probabilities(unsigned long long* result) {
    int tId = threadIdx.x + (blockIdx.x * blockDim.x);

    curandState state;
    curand_init((unsigned long long)clock() + tId, 0, 0, &state);

    //printf("%f\n%f\n", FIRST_SHOOTER_CHANCE, SECOND_SHOOTER_CHANCE);
    unsigned int first_shots = 0, second_shots = 0;
    while(true){
        ++first_shots;
        if (curand_uniform(&state) <= FIRST_SHOOTER_CHANCE) {
            break;
        }
    }
    while(true){
        ++second_shots;
        if (curand_uniform(&state) <= SECOND_SHOOTER_CHANCE) {
            break;
        }
    }
    
    if (first_shots > second_shots) {
        atomicAdd(result, 1);
    }
}
*/

/*
#define HEAD_RATE 0.58
#define ROLL_AMOUNT 10

__global__ void generate_movement_roll_probabilities(unsigned long long* result) {
    int tId = threadIdx.x + (blockIdx.x * blockDim.x);

    curandState state;
    curand_init((unsigned long long)clock() + tId, 0, 0, &state);

    short pos = 0;
    for (short i = 1; i <= ROLL_AMOUNT; ++i) {
        if (curand_uniform(&state) <= HEAD_RATE) {
            ++pos;
        }
        else {
            --pos;
        }
    }
    if (pos == 0) {
        atomicAdd(result, 1);
    }
}*/

#define URN1_WHITE_COUNT 13
#define URN1_BLACK_COUNT 33
#define URN2_WHITE_COUNT 33
#define URN2_BLACK_COUNT 7

__global__ void generate_urn_take_white_two_times(unsigned long long* result) {

    int tId = threadIdx.x + (blockIdx.x * blockDim.x);

    hiprandState state;
    hiprand_init((unsigned long long)clock() + tId, 0, 0, &state);
    if (hiprand_uniform(&state) <= 0.5) { //URN1
        //if (curand_uniform(&state) <= ((double) URN1_WHITE_COUNT / ((double) URN1_WHITE_COUNT + (double) URN1_BLACK_COUNT))) {
            if (hiprand_uniform(&state) >= ((double) URN1_WHITE_COUNT / ((double) URN1_WHITE_COUNT + (double) URN1_BLACK_COUNT))) {
                atomicAdd(result, 1);
                return;
            }
        //}
    }
    else {
        //if (curand_uniform(&state) <= ((double) URN2_WHITE_COUNT / ((double) URN2_WHITE_COUNT + (double) URN2_BLACK_COUNT) )) {
            if (hiprand_uniform(&state) >= ((double) URN2_WHITE_COUNT / ((double) URN2_WHITE_COUNT + (double) URN2_BLACK_COUNT) )) {
                atomicAdd(result, 1);
                return;
            }
        //}
    }

}


int main() {
    unsigned long long result = 0;
    unsigned long long *gpu_result;

    hipMalloc((void**)&gpu_result, sizeof(unsigned long long));

    printf("%f\n", ((double)URN1_WHITE_COUNT / ((double) URN1_WHITE_COUNT + (double) URN1_BLACK_COUNT)));

    generate_urn_take_white_two_times <<<ATTEMPT_COUNT, 1024>>> (gpu_result);

    hipMemcpy(&result, gpu_result, sizeof(unsigned long long), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    unsigned long long total = ATTEMPT_COUNT * (unsigned long long)1024;

    printf("%I64d\n", result);
    printf("%I64d\n", total);
    hipFree(gpu_result);

    return 0;
}